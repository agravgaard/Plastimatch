#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
#include "plm_config.h"

/****************************************************\
* Uncomment the line below to enable verbose output. *
* Enabling this should not nerf performance.         *
\****************************************************/
#define VERBOSE 1

/**********************************************************\
* Uncomment the line below to enable detailed performance  *
* reporting.  This measurement alters the system, however, *
* resulting in significantly slower kernel execution.      *
\**********************************************************/
#define TIME_KERNEL
#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

/*****************
*  C   #includes *
*****************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "drr_cuda.h"
#include "drr_cuda_p.h"
#include "drr_opts.h"
#include "file_util.h"
#include "plm_math.h"
#include "proj_image.h"
#include "volume.h"
#include "timer.h"


// P R O T O T Y P E S ////////////////////////////////////////////////////
void checkCUDAError(const char *msg);
__global__ void kernel_drr_i3 (float * dev_vol,  int2 img_dim, float2 ic, float3 nrm, float sad, float scale, float3 vol_offset, int3 vol_dim, float3 vol_pix_spacing);



// T E X T U R E S ////////////////////////////////////////////////////////
texture<float, 1, hipReadModeElementType> tex_img;
texture<float, 1, hipReadModeElementType> tex_matrix;
texture<float, 1, hipReadModeElementType> tex_coef;
texture<float, 3, hipReadModeElementType> tex_3Dvol;


// uses 3D textures and pre-calculated coefs to accelerate DRR generation.
void kernel_drr_i3 (float * dev_img, int2 img_dim, float2 ic, float3 nrm, float sad, float scale, float3 vol_offset, int3 vol_dim, float3 vol_pix_spacing)
{
    // CUDA 2.0 does not allow for a 3D grid, which severely
    // limits the manipulation of large 3D arrays of data.  The
    // following code is a hack to bypass this implementation
    // limitation.
    extern __shared__ float sdata[];
    float3 vp;
    int i,j,k;
    int x,y,xy7;
    float vol;

    unsigned int tid = threadIdx.x;

    x = blockIdx.x;
    y = blockIdx.y;
    xy7=7*(y*img_dim.x+x);
	
    if (abs(tex1Dfetch(tex_matrix, 5))>abs(tex1Dfetch(tex_matrix, 4))) {
	vp.x=vol_offset.x+threadIdx.x*vol_pix_spacing.x;
	vp.y=tex1Dfetch(tex_coef, xy7)*vp.x+tex1Dfetch(tex_coef, xy7+1);
	vp.z=tex1Dfetch(tex_coef, xy7+4)*vp.x
	    +tex1Dfetch(tex_coef, xy7+5)*vp.y+tex1Dfetch(tex_coef, xy7+6);

	i=  threadIdx.x;
	j=  __float2int_rd((vp.y-vol_offset.y)/vol_pix_spacing.y);
	k=  __float2int_rd((vp.z-vol_offset.z)/vol_pix_spacing.z);

	//if (j<0||j>=vol_dim.y||k<0||k>=vol_dim.z)
	if ((i-vol_dim.x/2)*(i-vol_dim.x/2)+(j-vol_dim.y/2)*(j-vol_dim.y/2)
	    > vol_dim.y*vol_dim.y/4||k<0||k>=vol_dim.z) 
	{
	    sdata[tid]=0.0f;
	} else {
	    vol=tex3D(tex_3Dvol,i,j,k);
	    sdata[tid]=(vol+1000.0f);
	}
    } else {
	vp.y=vol_offset.y+threadIdx.x*vol_pix_spacing.y;
	vp.x=tex1Dfetch(tex_coef, xy7+2)*vp.y+tex1Dfetch(tex_coef, xy7+3);
	vp.z=tex1Dfetch(tex_coef, xy7+4)*vp.x
	    +tex1Dfetch(tex_coef, xy7+5)*vp.y+tex1Dfetch(tex_coef, xy7+6);
	j=  threadIdx.x;
	i=  __float2int_rd((vp.x-vol_offset.x)/vol_pix_spacing.x);
	k=  __float2int_rd((vp.z-vol_offset.z)/vol_pix_spacing.z);

	if ((i-vol_dim.x/2)*(i-vol_dim.x/2)+(j-vol_dim.y/2)*(j-vol_dim.y/2)
	    > vol_dim.y*vol_dim.y/4||k<0||k>=vol_dim.z)
	{
	    sdata[tid]=0.0f;
	} else {
	    vol=tex3D(tex_3Dvol,i,j,k);
	    sdata[tid]=(vol+1000.0f);
	}
    }

    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>32; s>>=1) 
    {
        if (tid < s)
            sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        sdata[tid] += sdata[tid + 32]; EMUSYNC;
        sdata[tid] += sdata[tid + 16]; EMUSYNC;
        sdata[tid] += sdata[tid +  8]; EMUSYNC;
        sdata[tid] += sdata[tid +  4]; EMUSYNC;
        sdata[tid] += sdata[tid +  2]; EMUSYNC;
        sdata[tid] += sdata[tid +  1]; EMUSYNC;
    }

    // write result for this block to global mem
    if (tid == 0) 
	dev_img[blockIdx.x*img_dim.y + blockIdx.y] = sdata[0];
}

void*
drr_cuda_state_create (
    Proj_image *proj,
    Volume *vol,
    Drr_options *options
)
{
    Drr_cuda_state *state;
    Drr_kernel_args *kargs;

    state = (Drr_cuda_state *) malloc (sizeof(Drr_cuda_state));
    memset (state, 0, sizeof(Drr_cuda_state));

    state->kargs = kargs = (Drr_kernel_args*) malloc (sizeof(Drr_kernel_args));
    hipMalloc ((void**) &state->dev_matrix, 12 * sizeof(float));
    hipMalloc ((void**) &state->dev_kargs, sizeof(Drr_kernel_args));

    printf ("printf state = %p\n", state);
    printf ("printf state->kargs = %p\n", state->kargs);

    kargs->vol_offset.x = vol->offset[0];
    kargs->vol_offset.y = vol->offset[1];
    kargs->vol_offset.z = vol->offset[2];
    kargs->vol_dim.x = vol->dim[0];
    kargs->vol_dim.y = vol->dim[1];
    kargs->vol_dim.z = vol->dim[2];
    kargs->vol_pix_spacing.x = vol->pix_spacing[0];
    kargs->vol_pix_spacing.y = vol->pix_spacing[1];
    kargs->vol_pix_spacing.z = vol->pix_spacing[2];

    // prepare texture
    hipChannelFormatDesc ca_descriptor;
    hipExtent ca_extent;
    hipArray *dev_3Dvol=0;

    ca_descriptor = hipCreateChannelDesc<float>();
    ca_extent.width  = vol->dim[0];
    ca_extent.height = vol->dim[1];
    ca_extent.depth  = vol->dim[2];
    hipMalloc3DArray (&dev_3Dvol, &ca_descriptor, ca_extent);
    hipBindTextureToArray (tex_3Dvol, dev_3Dvol, ca_descriptor);

    hipMemcpy3DParms cpy_params = {0};
    cpy_params.extent   = ca_extent;
    cpy_params.kind     = hipMemcpyHostToDevice;
    cpy_params.dstArray = dev_3Dvol;

    //http://sites.google.com/site/cudaiap2009/cookbook-1#TOC-CUDA-3D-Texture-Example-Gerald-Dall
    // The pitched pointer is really tricky to get right. We give the
    // pitch of a row, then the number of elements in a row, then the
    // height, and we omit the 3rd dimension.
    cpy_params.srcPtr = make_hipPitchedPtr ((void*)vol->img, 
	ca_extent.width * sizeof(float), ca_extent.width , ca_extent.height);

    hipMemcpy3D (&cpy_params);

    hipMalloc ((void**) &state->dev_img, 
	options->image_resolution[0] * options->image_resolution[1] 
	* sizeof(float));

    hipMalloc ((void**) &state->dev_coef, 
	7 * options->image_resolution[0] * options->image_resolution[1] 
	* sizeof(float));
    checkCUDAError ("Unable to allocate coef devmem");
    state->host_coef = (float*) malloc (
	7 * options->image_resolution[0] * options->image_resolution[1] 
	* sizeof(float));
		
    return (void*) state;
}

void
drr_cuda_state_destroy (
    void *void_state
)
{
    Drr_cuda_state *state = (Drr_cuda_state*) void_state;
    
    hipFree (state->dev_img);
    hipFree (state->dev_kargs);
    hipFree (state->dev_matrix);
    hipFree (state->dev_coef);
    free (state->host_coef);
    free (state->kargs);
}

void
drr_cuda_render_volume_perspective (
    Proj_image *proj,
    void *void_state,
    Volume *vol, 
    double ps[2], 
    char *multispectral_fn, 
    Drr_options *options
)
{
    Timer timer, total_timer;
    double time_kernel = 0;
    double time_io = 0;
    int i;

    // CUDA device pointers
    Drr_cuda_state *state = (Drr_cuda_state*) void_state;
    Drr_kernel_args *kargs = state->kargs;
    float *host_coef = state->host_coef;

    // Start the timer
    plm_timer_start (&total_timer);
    plm_timer_start (&timer);

    // Load dynamic kernel arguments
    kargs->img_dim.x = proj->dim[0];
    kargs->img_dim.y = proj->dim[1];
    kargs->ic.x = proj->pmat->ic[0];
    kargs->ic.y = proj->pmat->ic[1];
    kargs->nrm.x = proj->pmat->nrm[0];
    kargs->nrm.y = proj->pmat->nrm[1];
    kargs->nrm.z = proj->pmat->nrm[2];
    kargs->sad = proj->pmat->sad;
    kargs->sid = proj->pmat->sid;
    for (i = 0; i < 12; i++) {
	kargs->matrix[i] = (float) proj->pmat->matrix[i];
    }

    // Precalculate coeff
    int xy7;
    double *matrix = proj->pmat->matrix;
    for (int x = 0; x < proj->dim[0] ; x++) {
	for (int y = 0; y < proj->dim[1] ; y++) {
	    xy7 = 7 * (y * proj->dim[0] + x);

#if defined (commentout)
	    host_coef[xy7]  =((y-ic[1])*proj->pmat->matrix[8]-proj->pmat->matrix[4])/(proj->pmat->matrix[5]-(y-ic[1])*proj->pmat->matrix[9]);
	    host_coef[xy7+2]=((y-ic[1])*proj->pmat->matrix[9]-proj->pmat->matrix[5])/(proj->pmat->matrix[4]-(y-ic[1])*proj->pmat->matrix[8]);
	    host_coef[xy7+1]=(y-ic[1])*proj->pmat->matrix[11]/(proj->pmat->matrix[5]-(y-ic[1])*proj->pmat->matrix[9]);
	    host_coef[xy7+3]=(y-ic[1])*proj->pmat->matrix[11]/(proj->pmat->matrix[4]-(y-ic[1])*proj->pmat->matrix[8]);
	    host_coef[xy7+4]=(x-ic[0])*proj->pmat->matrix[8]/proj->pmat->matrix[2];
	    host_coef[xy7+5]=(x-ic[0])*proj->pmat->matrix[9]/proj->pmat->matrix[2];
	    host_coef[xy7+6]=(x-ic[0])*proj->pmat->matrix[11]/proj->pmat->matrix[2];
#endif

	    host_coef[xy7]   = (y * matrix[8] - matrix[4])
		/ (matrix[5] - y * matrix[9]);
	    host_coef[xy7+2] = (y * matrix[9] - matrix[5])
		/ (matrix[4] - y * matrix[8]);
	    host_coef[xy7+1] = y * matrix[11]
		/ (matrix[5] - y * matrix[9]);
	    host_coef[xy7+3] = y * matrix[11]
		/ (matrix[4] - y * matrix[8]);
	    host_coef[xy7+4] = x * matrix[8] / matrix[2];
	    host_coef[xy7+5] = x * matrix[9] / matrix[2];
	    host_coef[xy7+6] = x * matrix[11] / matrix[2];
	}
    }

    time_io += plm_timer_report (&timer);
    plm_timer_start (&timer);

    hipMemcpy (state->dev_matrix, kargs->matrix, sizeof(kargs->matrix), 
	hipMemcpyHostToDevice);

    hipBindTexture (0, tex_matrix, state->dev_matrix, sizeof(kargs->matrix));

    hipMemcpy (state->dev_coef, host_coef, 
	7 * proj->dim[0] * proj->dim[1] * sizeof(float), 
	hipMemcpyHostToDevice);

    hipBindTexture (0, tex_coef, state->dev_coef, 
	7 * proj->dim[0] * proj->dim[1] * sizeof(float));

    // Thead Block Dimensions
    int tBlock_x = vol->dim[0];
    int tBlock_y = 1;
    int tBlock_z = 1;

    // Each element in the volume (each voxel) gets 1 thread
    int blocksInX = proj->dim[0];
    int blocksInY = proj->dim[1];
    dim3 dimGrid  = dim3(blocksInX, blocksInY);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

    // Invoke ze kernel  \(^_^)/
    // Note: proj->img AND proj->matrix are passed via texture memory

    int smemSize = vol->dim[0]  * sizeof(float);

    plm_timer_start (&timer);

    //-------------------------------------
    kernel_drr_i3<<< dimGrid, dimBlock, smemSize>>> (
	state->dev_img, 
	kargs->img_dim,
	kargs->ic,
	kargs->nrm,
	kargs->sad,
	kargs->scale,
	kargs->vol_offset,
	kargs->vol_dim,
	kargs->vol_pix_spacing);

    checkCUDAError("Kernel Panic!");

#if defined (TIME_KERNEL)
    // CUDA kernel calls are asynchronous...
    // In order to accurately time the kernel
    // execution time we need to set a thread
    // barrier here after its execution.
    hipDeviceSynchronize();
#endif

    time_kernel += plm_timer_report (&timer);

    // Unbind the image and projection matrix textures
    //hipUnbindTexture( tex_img );
    hipUnbindTexture (tex_matrix);
    hipUnbindTexture (tex_coef);

    // Copy reconstructed volume from device to host
    //hipMemcpy( vol->img, dev_vol, vol->npix * vol->pix_size, hipMemcpyDeviceToHost );
    hipMemcpy (proj->img, state->dev_img, 
	proj->dim[0] * proj->dim[1] * sizeof(float), 
	hipMemcpyDeviceToHost);
    checkCUDAError("Error: Unable to retrieve data volume.");
}


///////////////////////////////////////////////////////////////////////////
// FUNCTION: checkCUDAError() /////////////////////////////////////////////
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err) 
    {
        fprintf(stderr, "CUDA ERROR: %s (%s).\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
///////////////////////////////////////////////////////////////////////////



///////////////////////////////////////////////////////////////////////////
// Vim Editor Settings ////////////////////////////////////////////////////
// vim:ts=8:sw=8:cindent:nowrap
///////////////////////////////////////////////////////////////////////////
