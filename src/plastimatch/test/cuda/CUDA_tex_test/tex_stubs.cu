#include "hip/hip_runtime.h"
#include <stdio.h>
#include "tex_stubs.h"
#include "tex_kernels.h"

extern "C" void CUDA_texture_test(float* test_data, int elements)
{
	float* dev_test_data;
	float* dev_return;
	size_t test_size = elements * sizeof(float);

	// Allocate some global memory on the GPU
	hipMalloc((void**)&dev_test_data, test_size);
	checkCUDAError("hipMalloc(): dev_test_data"); 

	hipMalloc((void**)&dev_return, test_size);
	checkCUDAError("hipMalloc(): dev_return"); 

	// Copy test data to GPU global memory
	hipMemcpy(dev_test_data, test_data, test_size, hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy(): test_data -> dev_test_data"); 

	hipMemset(dev_return, 0, test_size);
	checkCUDAError("hipMemset(): dev_return"); 

	memset(test_data, 0, test_size);

	// Bind allocated global memory to texture reference
	hipBindTexture(0, tex_test, dev_test_data, test_size);
	checkCUDAError("hipBindTexture(): dev_test_data -> tex_test"); 

	// Define the execution configuration
	int threads_per_block = 128;
	int num_threads = elements;
	int num_blocks = (int)ceil(num_threads / (float)threads_per_block);

	dim3 dimGrid(num_blocks, 1, 1);
	dim3 dimBlock(threads_per_block, 1, 1);

	// Invoke the kernel
	kernel_texture<<<dimGrid, dimBlock>>>(dev_return, test_size);
	checkCUDAError("Kernel Panic!"); 

	// Copy results back
	hipMemcpy(test_data, dev_return, test_size, hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy(): dev_return -> test_data"); 

	// Cleanup
	hipUnbindTexture(tex_test);
	hipFree(dev_test_data);
	hipFree(dev_return);
}

extern "C" void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "\n\nCUDA ERROR: %s (%s).\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
